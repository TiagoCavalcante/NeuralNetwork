void destroy(
	const NeuralNetwork neuralNetwork
) {
	hipHostFree((float*)neuralNetwork.host_input);
	hipHostFree((float*)neuralNetwork.host_output);

	hipFree((float*)neuralNetwork.device_input);
	hipFree((float*)neuralNetwork.device_output);
	hipFree((float*)neuralNetwork.device_weightsInput);
	hipFree((float*)neuralNetwork.device_weightsHidden);
	hipFree((float*)neuralNetwork.device_activatedHiddenLayer);
	hipFree((float*)neuralNetwork.device_derivativeHiddenLayer);
	hipFree((float*)neuralNetwork.device_activatedOutputLayer);
	hipFree((float*)neuralNetwork.device_derivativeOutputLayer);
	hipFree((float*)neuralNetwork.device_hiddenBias);
	hipFree((float*)neuralNetwork.device_outputBias);

	hipStreamDestroy(neuralNetwork.stream);

	hipDeviceReset();
}