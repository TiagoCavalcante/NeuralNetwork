#include "hip/hip_runtime.h"
/**
 * initialize NeuralNetwork struct and allocate all variables on GPU memory
 * @param cudaDevice - GPU number, 1st GPU is 0
 * @returns a pointer to the new NeuralNetowrk structure or 0 if it fails
*/
NeuralNetwork init(
	const unsigned short inputNodes,
	const unsigned short hiddenNodes,
	const unsigned short outputNodes,
	float learningRate,
	unsigned char cudaDevice,
	const unsigned short threadsPerBlock
) {
	if (hipSetDevice(cudaDevice) != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");

		return {};
	}

	dim3 threadsPerBlock2d(threadsPerBlock, threadsPerBlock);

	dim3 weightsInputBlocksPerGrid(
		(unsigned short)ceil((float)hiddenNodes / (float)threadsPerBlock),
		(unsigned short)ceil((float)inputNodes  / (float)threadsPerBlock)
	);
	dim3 weightsHiddenBlocksPerGrid(
		(unsigned short)ceil((float)outputNodes / (float)threadsPerBlock),
		(unsigned short)ceil((float)hiddenNodes / (float)threadsPerBlock)
	);

	float* host_input;
	float* host_output;

	float* device_input;
	float* device_output;
	float* device_weightsInput;
	float* device_weightsHidden;
	float* device_activatedHiddenLayer;
	float* device_derivativeHiddenLayer;
	float* device_activatedOutputLayer;
	float* device_derivativeOutputLayer;
	float* device_hiddenBias;
	float* device_outputBias;

	if (hipHostMalloc(
		&host_input,
		inputNodes * sizeof(float)
	) != hipSuccess) goto MallocError;

	if (hipHostMalloc(
		&host_output,
		outputNodes * sizeof(float)
	) != hipSuccess) goto MallocError;

	if (hipMalloc(
		&device_input,
		inputNodes * sizeof(float)
	) != hipSuccess) goto MallocError;

	if (hipMalloc(
		&device_output,
		outputNodes * sizeof(float)
	) != hipSuccess) goto MallocError;

	if (hipMalloc(
		&device_weightsInput,
		hiddenNodes * inputNodes * sizeof(float)
	) != hipSuccess) goto MallocError;

	if (hipMalloc(
		&device_weightsHidden,
		outputNodes * hiddenNodes * sizeof(float)
	) != hipSuccess) goto MallocError;

	if (hipMalloc(
		&device_activatedHiddenLayer,
		hiddenNodes * sizeof(float)
	) != hipSuccess) goto MallocError;

	if (hipMalloc(
		&device_derivativeHiddenLayer,
		hiddenNodes * sizeof(float)
	) != hipSuccess) goto MallocError;

	if (hipMalloc(
		&device_activatedOutputLayer,
		outputNodes * sizeof(float)
	) != hipSuccess) goto MallocError;

	if (hipMalloc(
		&device_derivativeOutputLayer,
		outputNodes * sizeof(float)
	) != hipSuccess) goto MallocError;

	if (hipMalloc(
		&device_hiddenBias,
		hiddenNodes * sizeof(float)
	) != hipSuccess) goto MallocError;

	if (hipMalloc(
		&device_outputBias,
		outputNodes * sizeof(float)
	) != hipSuccess) goto MallocError;

	hipStream_t stream;
	hipStreamCreate(&stream);

	fillWeightsAndBiases(
		device_weightsInput,
		device_weightsHidden,
		device_hiddenBias,
		device_outputBias,
		inputNodes,
		hiddenNodes,
		outputNodes
	);

	return {
		inputNodes,
		hiddenNodes,
		outputNodes,

		learningRate,

		stream,

		threadsPerBlock,
		threadsPerBlock2d,

		// hiddenLayerBlocksPerGrid
		(unsigned short)ceil((float)hiddenNodes / (float)threadsPerBlock),
		// outputBlocksPerGrid
		(unsigned short)ceil((float)outputNodes / (float)threadsPerBlock),

		weightsInputBlocksPerGrid,
		weightsHiddenBlocksPerGrid,

		host_input,
		host_output,

		device_input,
		device_output,
		device_weightsInput,
		device_weightsHidden,
		device_activatedHiddenLayer,
		device_derivativeHiddenLayer,
		device_activatedOutputLayer,
		device_derivativeOutputLayer,
		device_hiddenBias,
		device_outputBias
	};

MallocError:
	fprintf(stderr, "Memory is leaking!");

	hipHostFree(host_input);
	hipHostFree(host_output);

	hipFree(device_input);
	hipFree(device_output);
	hipFree(device_weightsInput);
	hipFree(device_weightsHidden);
	hipFree(device_activatedHiddenLayer);
	hipFree(device_derivativeHiddenLayer);
	hipFree(device_activatedOutputLayer);
	hipFree(device_derivativeOutputLayer);
	hipFree(device_hiddenBias);
	hipFree(device_outputBias);

	return {};
}