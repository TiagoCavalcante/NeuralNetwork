#include "hip/hip_runtime.h"
__global__ void activateAndDeriveMatrixByVectorMultiplication(
	float* __restrict__ resultVector,
	float* __restrict__ resultVectorDerivative,
	const float* __restrict__ vector,
	const float* __restrict__ matrix,
	const float* __restrict__ bias,
	const unsigned short matrixRows,
	const unsigned short matrixColumns
) {
	const unsigned short elementNumber = blockIdx.x * blockDim.x + threadIdx.x;

	if(elementNumber < matrixRows) {
		float sum = 0;

		for (unsigned short i = 0; i < matrixColumns; ++i) {
			sum += vector[i] * matrix[elementNumber * matrixColumns + i];
		}

		resultVector[elementNumber] = sigmoid(sum + bias[elementNumber]);
		resultVectorDerivative[elementNumber] = sigmoidDerivative(sum);
	}
}